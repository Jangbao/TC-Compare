#include "hip/hip_runtime.h"
#include "../include/gpurun.cuh"
#include <hip/hip_runtime_api.h>
template <typename T>
void singleParamTestGPURun(Param param)
{
  T *offsetVector;
  T *indexVector;
  T vertexCount;
  T edgeCount;

  hipProfilerStop();
  {
    thrust::device_vector<int> memory(1);
  }

  readBinGraph(param.fileName, offsetVector, indexVector,
               vertexCount, edgeCount);

  std::cout << "dataset\t" << param.fileName << std::endl;
  std::cout << "Number of nodes: " << vertexCount
            << ", number of edges: " << edgeCount << std::endl;

  // std::chrono::time_point<std::chrono::system_clock> execStart, execEnd, kernelStart, kernelEnd, memAllocEnd;

  // execStart = std::chrono::system_clock::now();
  hipProfilerStop();
  thrust::device_vector<T> dOffsetVector(offsetVector,
                                         offsetVector + vertexCount + 1);
  thrust::device_vector<T> dIndexVector(indexVector, indexVector + edgeCount);
  thrust::device_vector<T> dTriangleOutputVector(vertexCount, 0);

  T const *const dOffset = thrust::raw_pointer_cast(dOffsetVector.data());
  T const *const dIndex = thrust::raw_pointer_cast(dIndexVector.data());
  T *const dTriangle = thrust::raw_pointer_cast(dTriangleOutputVector.data());
  hipDeviceSynchronize();
  // memAllocEnd = std::chrono::system_clock::now();

  unsigned int blocks = param.blocks;
  blocks = 1000000;
  if (edgeCount / 10 < blocks)
  {
    blocks = edgeCount / 10;
  }
  unsigned int blockSize = param.threadCount;
  T threadsPerIntsctn = param.threadPerInt;
  T intsctnPerBlock = param.threadCount / param.threadPerInt;
  T threadShift = std::log2(param.threadPerInt);
  T triangleCount;

  double total_kernel_use = 0;
  int iterator_count = param.blocks;
  double startKernel, ee;
  for (int i = 0; i < iterator_count; i++)
  {
    startKernel = omp_get_wtime();
    hipProfilerStart();
    kernelCall(blocks, blockSize, vertexCount, dOffset,
               dIndex, dTriangle, threadsPerIntsctn, intsctnPerBlock, threadShift);
    hipDeviceSynchronize();
    hipProfilerStop();
    triangleCount = thrust::reduce(dTriangleOutputVector.begin(),
                                   dTriangleOutputVector.end());
    ee = omp_get_wtime();
    total_kernel_use += ee - startKernel;
  }
  // std::chrono::duration<float, std::milli> memAllocDuration = memAllocEnd -
  //                                                             execStart;
  // std::chrono::duration<float, std::milli> tCountDuration = execEnd -
  //                                                           memAllocEnd;
  // std::chrono::duration<float, std::milli> kernelDuration = kernelEnd -
  //                                                           kernelStart;
  // std::chrono::duration<float, std::milli> execDuration = execEnd -
  //                                                         execStart;

  printf("iter %d, avg kernel use %lf s\n", iterator_count, total_kernel_use / iterator_count);
  printf("triangle count %ld \n\n", triangleCount);

  // std::cout << "vertexCount\t"
  //           << "edgeCount \t"
  //           << "totalTriangleCount\t"
  //           << "memAlloc\t"
  //           << "tCount\t"
  //           << "kernel\t"
  //           << "exec\n";

  // std::cout << vertexCount << "\t" << edgeCount << "\t" << totalTriangleCount << "\t" << memAllocDuration.count() << "ms\t" << tCountDuration.count() << "ms\t" << kernelDuration.count() << "ms\t" << execDuration.count() << "ms\n";

  delete[] offsetVector;
  delete[] indexVector;
}

template <typename T>
void singleGPURun(Param param,
                  T *offsetVector, T vertexCount, T *indexVector, T edgeCount)
{
  {
    thrust::device_vector<int> memory(1);
  }
  std::string fileName = std::string("runresult/") + param.fileName +
                         std::string(".o.") + std::to_string(param.blocks) + std::string(".") +
                         std::to_string(param.threadCount) + std::string(".") +
                         std::to_string(param.threadPerInt);
  std::ofstream fout(fileName, std::ios::out | std::ios::app);

  std::chrono::time_point<std::chrono::system_clock> execStart, execEnd,
      memAllocEnd;
  execStart = std::chrono::system_clock::now();
  thrust::device_vector<T> dOffsetVector(offsetVector,
                                         offsetVector + vertexCount + 1);
  thrust::device_vector<T> dIndexVector(indexVector, indexVector + edgeCount);
  thrust::device_vector<T> dTriangleOutputVector(vertexCount, 0);

  T const *const dOffset = thrust::raw_pointer_cast(dOffsetVector.data());
  T const *const dIndex = thrust::raw_pointer_cast(dIndexVector.data());
  T *const dTriangle = thrust::raw_pointer_cast(dTriangleOutputVector.data());
  hipDeviceSynchronize();
  memAllocEnd = std::chrono::system_clock::now();

  unsigned int blocks = param.blocks;
  unsigned int blockSize = param.threadCount;
  T threadsPerIntsctn = param.threadPerInt;
  T intsctnPerBlock = param.threadCount / param.threadPerInt;
  T threadShift = std::log2(param.threadPerInt);
  kernelCall(blocks, blockSize, vertexCount, dOffset,
             dIndex, dTriangle, threadsPerIntsctn, intsctnPerBlock, threadShift);
  hipDeviceSynchronize();
  execEnd = std::chrono::system_clock::now();
  T totalTriangleCount = thrust::reduce(dTriangleOutputVector.begin(),
                                        dTriangleOutputVector.end());

  std::chrono::duration<float, std::milli> memAllocDuration = memAllocEnd -
                                                              execStart;
  std::chrono::duration<float, std::milli> tCountDuration = execEnd -
                                                            memAllocEnd;
  std::chrono::duration<float, std::milli> execDuration = execEnd -
                                                          execStart;
  fout << "ctime\t1\t" << tCountDuration.count() << "\n\n";
  fout.close();
  /*
  std::cout<<vertexCount<<"\t"<<totalTriangleCount<<"\t"<<
    memAllocDuration.count()<<"\t"<<tCountDuration.count()<<"\t"<<
    execDuration.count()<<"\n";
    */
}

template <typename T>
void allParamTestGPURun(Param param)
{
  T *offsetVector;
  T *indexVector;
  T vertexCount;
  T edgeCount;

  {
    thrust::device_vector<int> memory(1);
  }

  readBinGraph(param.fileName, offsetVector, indexVector,
               vertexCount, edgeCount);
  hipDeviceSynchronize();

  thrust::device_vector<T> dOffsetVector(offsetVector, offsetVector + vertexCount + 1);
  thrust::device_vector<T> dIndexVector(indexVector, indexVector + edgeCount);
  thrust::device_vector<T> dTriangleOutputVector(dOffsetVector.size(), 0);

  T const *const dOffset = thrust::raw_pointer_cast(dOffsetVector.data());
  T const *const dIndex = thrust::raw_pointer_cast(dIndexVector.data());
  T *const dTriangle = thrust::raw_pointer_cast(dTriangleOutputVector.data());

  std::string dataset = getFileName(param.fileName);
  std::string fileOutName = std::string("./output/") + dataset + std::string("_") + std::to_string(param.blocks) + std::string(".output");

  printf("output: %s\n", fileOutName.c_str());
  std::ofstream writeFile(fileOutName);
  writeFile << param.fileName << std::endl;
  writeFile << "Number of nodes: " << vertexCount
            << ", number of edges: " << edgeCount << std::endl;
  T sumTriangles;
  for (auto paramBlockSize : globalParam::blockSizeParam)
  {
    for (auto paramThreadsPerIntsctn : globalParam::threadPerIntersectionParam)
    {
      double total_kernel_use = 0;
      int iterator_count = param.blocks;
      double startKernel, ee;
      for (int i = 0; i < iterator_count; i++)
      {
        startKernel = omp_get_wtime();

        thrust::fill(dTriangleOutputVector.begin(), dTriangleOutputVector.end(), 0);
        unsigned int blocks = param.blocks;
        unsigned int blockSize = paramBlockSize;
        T threadsPerIntsctn = paramThreadsPerIntsctn;
        T intsctnPerBlock = paramBlockSize / paramThreadsPerIntsctn;
        T threadShift = std::log2(paramThreadsPerIntsctn);
        kernelCall(blocks, blockSize, vertexCount, dOffset,
                   dIndex, dTriangle, threadsPerIntsctn, intsctnPerBlock, threadShift);
        sumTriangles = thrust::reduce(dTriangleOutputVector.begin(), dTriangleOutputVector.end());
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
          printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        }

        ee = omp_get_wtime();
        total_kernel_use += ee - startKernel;
      }
      writeFile << "block size " << paramBlockSize << ", threads per intersection " << paramThreadsPerIntsctn << std::endl;
      writeFile << "iter " << iterator_count << ", avg kernel use " << total_kernel_use / iterator_count << " s" << std::endl;
      writeFile << "triangle count  " << sumTriangles << std::endl
                << std::endl;
    }
  }
  writeFile.close();
}

template void singleParamTestGPURun<int32_t>(Param param);

template void singleGPURun<int32_t>(Param param,
                                    int32_t *offsetVector, int32_t vertexCount,
                                    int32_t *indexVector, int32_t edgeCount);
template void allParamTestGPURun<int32_t>(Param param);

template void singleParamTestGPURun<int64_t>(Param param);
template void singleGPURun<int64_t>(Param param,
                                    int64_t *offsetVector, int64_t vertexCount,
                                    int64_t *indexVector, int64_t edgeCount);

template void allParamTestGPURun<int64_t>(Param param);
