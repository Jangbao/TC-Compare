#include "../include/main.cuh"

template <typename T>
void run(int argc, char *argv[])
{
  Param param = paramProcess(argc, argv);
  if (!param.valid)
  {
    std::cerr << "Exiting..\n";
    exit(0);
  }

  hipSetDevice(param.device);
  // int dev = param.device;
  // int processor_count;
  // hipGetDevice(&dev);
  // hipDeviceGetAttribute(&processor_count, hipDeviceAttributeMultiprocessorCount, dev);
  // param.blocks = processor_count * 8;

  if (!param.testAll)
  {
    singleParamTestGPURun<T>(param);
  }
  else
  {
    allParamTestGPURun<T>(param);
  }

  // if((param.deviceMap).size() > 0)
  // {
  //   hipSetDevice((param.deviceMap).at(0));
  //   std::cerr<<"\ndeviceMap provided\n";
  // }
  // else
  // {
  //   hipSetDevice(0);
  // }
}

int main(int argc, char *argv[])
{
  run<int64_t>(argc, argv);
  return 0;
}
